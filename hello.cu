
#include <hip/hip_runtime.h>
#include "stdio.h"
#include "omp.h"

__global__ void cuda_hello(){
    printf("Hello World from GPU!\n");
}

int main() {
    int tid, nthreads;

    cuda_hello<<<1,1>>>(); 

    #pragma omp parallel private(tid, nthreads)
    {
        tid = omp_get_thread_num();
        nthreads = omp_get_num_threads();

        printf("Greetings from thread %i of %i\n", tid, nthreads);
    }

    hipDeviceSynchronize();
    return 0;
}